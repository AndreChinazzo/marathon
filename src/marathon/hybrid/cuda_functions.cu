#include "../../../include/marathon/exceptions.h"

#include <cublasXt.h>
#include "stdio.h"

namespace marathon {

namespace hybrid {

namespace cuda {

cublasXtHandle_t cublasXtHandle;

extern "C" void initCublasXt() {
	cublasXtCreate(&cublasXtHandle);

	// use first gpu
	// TODO: support multi gpu usage
	int devices[] = { 0 };
	int numDevices = 1;

	cublasXtDeviceSelect(cublasXtHandle, numDevices, devices);

	// set block dimension
	// TODO: add logic to choose optimal value
	//cublasXtSetBlockDim(cublasXtHandle, 6000);
}

extern "C" void finalizeCublasXt() {
	cublasXtDestroy(cublasXtHandle);
}

extern "C" void multFloatXt(const float* A, const size_t ldA, const float* B,
		const size_t ldB, float* C, const size_t ldC, const size_t n) {
	hipblasStatus_t err;

	const float alpha_d = 1.0;
	const float beta_d = 0.0;

	err = cublasXtSgemm(cublasXtHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
			&alpha_d, B, ldB, A, ldA, &beta_d, C, ldC);

	if (err != HIPBLAS_STATUS_SUCCESS) {
		throw CUBLAS_EXCEPTION;
	}
}

extern "C" void multDoubleXt(const double* A, const size_t ldA, const double* B,
		const size_t ldB, double* C, const size_t ldC, const size_t n) {

	hipblasStatus_t err;

	const double alpha_d = 1.0;
	const double beta_d = 0.0;

	err = cublasXtDgemm(cublasXtHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
			&alpha_d, B, ldB, A, ldA, &beta_d, C, ldC);

	if (err != HIPBLAS_STATUS_SUCCESS) {
		throw CUBLAS_EXCEPTION;
	}
}

}

}

}
