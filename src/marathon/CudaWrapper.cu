
#include <hip/hip_runtime.h>
/*
 * CudaWrapper.cu
 *
 *  Created on: Mar 23, 2016
 *      Author: rechner
 */

namespace marathon {

extern void myCudaMallocPitch(void** devPtr, size_t* pitch, size_t width,
		size_t height) {
	hipMallocPitch(devPtr, pitch, width, height);
}

extern void myCudaMalloc(void** devPtr, size_t size) {
	hipMalloc(devPtr, size);
}

extern void myCudaFree(void* devPtr) {
	hipFree(devPtr);
}

extern void myCudaMemcpyHostToDevice(void* dst, const void * src,
		size_t count) {
	hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

extern void myCudaMemset2D(void* devPtr, size_t pitch, int value, size_t width,
		size_t height) {
	hipMemset2D(devPtr, pitch, value, width, height);
}

extern void myCudaMemcpy2DHostToDevice(void* dst, size_t dpitch,
		const void* src, size_t spitch, size_t width, size_t height) {
	hipMemcpy2D(dst, dpitch, src, spitch, width, height,
			hipMemcpyHostToDevice);
}

extern void myCudaMemcpy2DDeviceToHost(void* dst, size_t dpitch,
		const void* src, size_t spitch, size_t width, size_t height) {
	hipMemcpy2D(dst, dpitch, src, spitch, width, height,
			hipMemcpyDeviceToHost);
}

extern void myCudaMemcpy2DDeviceToDevice(void* dst, size_t dpitch,
		const void* src, size_t spitch, size_t width, size_t height) {
	hipMemcpy2D(dst, dpitch, src, spitch, width, height,
			hipMemcpyDeviceToDevice);
}

}

