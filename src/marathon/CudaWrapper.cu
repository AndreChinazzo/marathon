
#include <hip/hip_runtime.h>
/*
 * CudaWrapper.cu
 *
 * Created on: Mar 23, 2016
 * Author: Steffen Rechner <steffen.rechner@informatik.uni-halle.de>
 *
 * This file is part of the marathon software.
 *
 * Copyright (c) 2016, Steffen Rechner
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is furnished
 * to do so, subject to the following conditions:
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

namespace marathon {

namespace cuda {

extern void myCudaMallocPitch(void** devPtr, size_t* pitch, size_t width,
		size_t height) {
	hipMallocPitch(devPtr, pitch, width, height);
}

extern void myCudaMalloc(void** devPtr, size_t size) {
	hipMalloc(devPtr, size);
}

extern void myCudaFree(void* devPtr) {
	hipFree(devPtr);
}

extern void myCudaMemcpyHostToDevice(void* dst, const void * src,
		size_t count) {
	hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

extern void myCudaMemset2D(void* devPtr, size_t pitch, int value, size_t width,
		size_t height) {
	hipMemset2D(devPtr, pitch, value, width, height);
}

extern void myCudaMemcpy2DHostToDevice(void* dst, size_t dpitch,
		const void* src, size_t spitch, size_t width, size_t height) {
	hipMemcpy2D(dst, dpitch, src, spitch, width, height,
			hipMemcpyHostToDevice);
}

extern void myCudaMemcpy2DDeviceToHost(void* dst, size_t dpitch,
		const void* src, size_t spitch, size_t width, size_t height) {
	hipMemcpy2D(dst, dpitch, src, spitch, width, height,
			hipMemcpyDeviceToHost);
}

extern void myCudaMemcpy2DDeviceToDevice(void* dst, size_t dpitch,
		const void* src, size_t spitch, size_t width, size_t height) {
	hipMemcpy2D(dst, dpitch, src, spitch, width, height,
			hipMemcpyDeviceToDevice);
}

}
}

