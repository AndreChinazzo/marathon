/*
 * CudaInitFinalize.cpp
 *
 *  Created on: Mar 23, 2016
 *      Author: rechner
 */


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cublasXt.h>

namespace marathon {

// a handle for using cublas library
hipblasHandle_t cublasHandle;
cublasXtHandle_t cublasXtHandle;

extern void cudaInit() {

	// check of cuda capable gpu is available
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess || deviceCount == 0)
		return;

	hipblasCreate(&cublasHandle);

	cublasXtCreate(&cublasXtHandle);

	// use first gpu
	// TODO: support multi gpu usage
	int devices[] = { 0 };
	int numDevices = 1;

	cublasXtDeviceSelect(cublasXtHandle, numDevices, devices);

}

extern void cudaFinalize() {
	hipblasDestroy(cublasHandle);
	cublasXtDestroy(cublasXtHandle);
}

}

