/*
 * CudaInitFinalize.cpp
 *
 *  Created on: Mar 23, 2016
 *      Author: rechner
 */


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cublasXt.h>

namespace marathon {

// a handle for using cublas library
hipblasHandle_t cublasHandle;
cublasXtHandle_t cublasXtHandle;

extern bool cudaInit() {

	// check of cuda capable gpu is available
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess || deviceCount == 0)
		return false;

	hipblasCreate(&cublasHandle);

	cublasXtCreate(&cublasXtHandle);

	// use first gpu
	// TODO: support multi gpu usage
	int devices[] = { 0 };
	int numDevices = 1;

	cublasXtDeviceSelect(cublasXtHandle, numDevices, devices);

	return true;
}

extern void cudaFinalize() {

	hipblasDestroy(cublasHandle);
	cublasXtDestroy(cublasXtHandle);
	}

}

